#include "hip/hip_runtime.h"
// Chapter 10. Reduction

#include "tests.cuh"

#include <hip/hip_runtime.h>
#include <>

#define CONTENT_TYPE  int32_t
#define CONTENT_COUNT 100000

__global__ void kernel_reduction(
	CONTENT_TYPE* content, int32_t totalCount,
	CONTENT_TYPE* outResult)
{
	uint32_t ix = threadIdx.x + (2 * blockDim.x * blockIdx.x);

	if (ix >= totalCount) {
		return;
	}

	for (uint32_t stride = blockDim.x; stride >= 1; stride /= 2) {
		if (threadIdx.x < stride && ix + stride < totalCount) {
			content[ix] += content[ix + stride];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		atomicAdd(outResult, content[(2 * blockDim.x * blockIdx.x)]);
	}
}

int runTest_reduction(int argc, char* argv[])
{
	// ------------------------------------------
	// Query device properties

	int cudaDeviceId;
	CUDA_ASSERT(hipGetDevice(&cudaDeviceId));

	hipDeviceProp_t deviceProps;
	CUDA_ASSERT(hipGetDeviceProperties(&deviceProps, cudaDeviceId));

	const float KHZ_TO_GHZ = 0.001f * 0.001f;

	puts("CUDA device properties");
	// CUDA gives you all these info!?
	printf("\ttotalConstMem      : %zu bytes\n", deviceProps.totalConstMem);
	printf("\tsharedMemPerBlock  : %zu bytes\n", deviceProps.sharedMemPerBlock);
	printf("\twarpSize           : %d\n", deviceProps.warpSize);
	printf("\tclockRate          : %f GHz\n", KHZ_TO_GHZ * (float)deviceProps.clockRate);
	printf("\tmemoryBusWidth     : %d bits\n", deviceProps.memoryBusWidth);
	printf("\tmemoryClockRate    : %f GHz\n", KHZ_TO_GHZ * deviceProps.memoryClockRate);
	printf("\tmaxThreadsPerBlock : %d\n", deviceProps.maxThreadsPerBlock);

	// ------------------------------------------
	// Host -> device

	std::vector<CONTENT_TYPE> content(CONTENT_COUNT, 0);
	for (size_t i = 0; i < CONTENT_COUNT; ++i) {
		content[i] = i + 1;
	}

	const size_t contentTotalBytes = sizeof(CONTENT_TYPE) * CONTENT_COUNT;

	CONTENT_TYPE* content_dev;
	CONTENT_TYPE* result_dev;
	CUDA_ASSERT(hipMalloc(&content_dev, contentTotalBytes));
	CUDA_ASSERT(hipMalloc(&result_dev, sizeof(CONTENT_TYPE)));
	CUDA_ASSERT(hipMemcpy(content_dev, content.data(), contentTotalBytes, hipMemcpyHostToDevice));
	CUDA_ASSERT(hipMemset(result_dev, 0, sizeof(CONTENT_TYPE)));

	// ------------------------------------------
	// Kernel: reduction (sum)

	puts("Run kernel: reduction (sum)");

	const dim3 blockDim(deviceProps.maxThreadsPerBlock, 1, 1);
	const dim3 numBlocks = calcNumBlocks(dim3(CONTENT_COUNT / 2, 1, 1), blockDim);

	kernel_reduction<<<numBlocks, blockDim>>>(
		content_dev, CONTENT_COUNT,
		result_dev);

	// ------------------------------------------
	// Device -> host

	CONTENT_TYPE result;
	CUDA_ASSERT(hipMemcpy(&result, result_dev, sizeof(CONTENT_TYPE), hipMemcpyDeviceToHost));

	puts("Compare results...");
	{
		CONTENT_TYPE answer = 0;
		for (CONTENT_TYPE x : content) {
			answer += x;
		}
		assert(answer == result);
	}

	return 0;
}
