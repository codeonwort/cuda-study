#include "hip/hip_runtime.h"
// Chapter 11. Prefix sum (scan)

#include "tests.cuh"

#include <hip/hip_runtime.h>
#include <>

#include <random>

#define CONTENT_TYPE  int32_t
// TODO: Only works if CONTENT_COUNT <= BLOCK_DIM
#define CONTENT_COUNT 1000
#define BLOCK_DIM     1024

// TODO: Double buffering
__global__ void kernel_scan_Kogge_Stone(
	CONTENT_TYPE* src, uint32_t totalCount,
	CONTENT_TYPE* dst)
{
	__shared__ CONTENT_TYPE XY[BLOCK_DIM];

	uint32_t ix = (blockIdx.x * blockDim.x) + threadIdx.x;
	uint32_t tid = threadIdx.x;
	if (ix < totalCount) {
		XY[tid] = src[ix];
	} else {
		XY[tid] = CONTENT_TYPE(0);
	}

	for (uint32_t stride = 1; stride < blockDim.x; stride *= 2) {
		__syncthreads();
		CONTENT_TYPE temp;
		if (tid >= stride) {
			temp = XY[tid] + XY[tid - stride];
		}
		__syncthreads();
		if (tid >= stride) {
			XY[tid] = temp;
		}
	}
	if (ix < totalCount) {
		dst[ix] = XY[tid];
	}
}

#define SECTION_SIZE  2048
__global__ void kernel_scan_Brent_Kung(
	CONTENT_TYPE* src, uint32_t totalCount,
	CONTENT_TYPE* dst)
{
	__shared__ CONTENT_TYPE XY[SECTION_SIZE];

	uint32_t ix = (2 * blockIdx.x * blockDim.x) + threadIdx.x;
	uint32_t tid = threadIdx.x;
	if (ix < totalCount) {
		XY[tid] = src[ix];
	}
	if (ix + blockDim.x < totalCount) {
		XY[tid + blockDim.x] = src[ix + blockDim.x];
	}

	// Reduction tree phase
	for (uint32_t stride = 1; stride < blockDim.x; stride *= 2) {
		__syncthreads();
#if 1
		// Map a continuous section of threads to the XY positions
		// whose indices are of the form k*2^n - 1
		uint32_t p = (tid + 1) * 2 * stride - 1;
		if (p < SECTION_SIZE) {
			XY[p] += XY[p - stride];
		}
#else
		// Causes significant control divergence
		if ((tid + 1) % (2 * stride) == 0) {
			XY[tid] += XY[tid - stride];
		}
#endif
	}
	// Reverse tree phase
	for (uint32_t stride = SECTION_SIZE / 4; stride > 0; stride /= 2) {
		__syncthreads();
		uint32_t p = (tid + 1) * stride * 2 - 1;
		if (p + stride < SECTION_SIZE) {
			XY[p + stride] += XY[p];
		}
	}
	__syncthreads();
	if (ix < totalCount) {
		dst[ix] = XY[tid];
	}
	if (ix + blockDim.x < totalCount) {
		dst[ix + blockDim.x] = XY[tid + blockDim.x];
	}
}

int runTest_scan(int argc, char* argv[])
{
	// ------------------------------------------
	// Query device properties

	int cudaDeviceId;
	CUDA_ASSERT(hipGetDevice(&cudaDeviceId));

	hipDeviceProp_t deviceProps;
	CUDA_ASSERT(hipGetDeviceProperties(&deviceProps, cudaDeviceId));

	const float KHZ_TO_GHZ = 0.001f * 0.001f;

	puts("CUDA device properties");
	// CUDA gives you all these info!?
	printf("\ttotalConstMem      : %zu bytes\n", deviceProps.totalConstMem);
	printf("\tsharedMemPerBlock  : %zu bytes\n", deviceProps.sharedMemPerBlock);
	printf("\twarpSize           : %d\n", deviceProps.warpSize);
	printf("\tclockRate          : %f GHz\n", KHZ_TO_GHZ * (float)deviceProps.clockRate);
	printf("\tmemoryBusWidth     : %d bits\n", deviceProps.memoryBusWidth);
	printf("\tmemoryClockRate    : %f GHz\n", KHZ_TO_GHZ * deviceProps.memoryClockRate);
	printf("\tmaxThreadsPerBlock : %d\n", deviceProps.maxThreadsPerBlock);

	assert(deviceProps.maxThreadsPerBlock >= BLOCK_DIM);

	// ------------------------------------------
	// Host -> device

	std::random_device randDevice;
	std::mt19937 randGen(randDevice());
	//std::uniform_real_distribution<CONTENT_TYPE> randSampler(-1.0f, 1.0f);
	std::uniform_int_distribution<CONTENT_TYPE> randSampler(-128, 128);

	std::vector<CONTENT_TYPE> input(CONTENT_COUNT);
	for (size_t i = 0; i < CONTENT_COUNT; ++i) {
		CONTENT_TYPE x = randSampler(randGen);
		input[i] = x;
	}
	printf("Generate %d input data\n", CONTENT_COUNT);

	std::vector<CONTENT_TYPE> answer(CONTENT_COUNT);
	answer[0] = input[0];
	for (size_t i = 1; i < CONTENT_COUNT; ++i) {
		answer[i] = answer[i - 1] + input[i];
	}

	const size_t contentTotalBytes = sizeof(CONTENT_TYPE) * CONTENT_COUNT;

	CONTENT_TYPE* content_dev;
	CONTENT_TYPE* content2_dev;
	CONTENT_TYPE* result_dev;
	CONTENT_TYPE* result2_dev;
	CUDA_ASSERT(hipMalloc(&content_dev, contentTotalBytes));
	CUDA_ASSERT(hipMalloc(&content2_dev, contentTotalBytes));
	CUDA_ASSERT(hipMalloc(&result_dev, contentTotalBytes));
	CUDA_ASSERT(hipMalloc(&result2_dev, contentTotalBytes));
	CUDA_ASSERT(hipMemcpy(content_dev, input.data(), contentTotalBytes, hipMemcpyHostToDevice));
	CUDA_ASSERT(hipMemcpy(content2_dev, input.data(), contentTotalBytes, hipMemcpyHostToDevice));

	// ------------------------------------------
	// Kernel: Prefix sum (scan)

	const dim3 blockDim(BLOCK_DIM, 1, 1);
	const dim3 numBlocks = calcNumBlocks(dim3(CONTENT_COUNT, 1, 1), BLOCK_DIM);
	kernel_scan_Kogge_Stone<<<numBlocks, blockDim>>>(
		content_dev, CONTENT_COUNT,
		result_dev);

	const dim3 numBlocks2 = calcNumBlocks(dim3(CONTENT_COUNT / 2, 1, 1), BLOCK_DIM);
	kernel_scan_Brent_Kung<<<numBlocks2, blockDim>>>(
		content2_dev, CONTENT_COUNT,
		result2_dev);

	// ------------------------------------------
	// Device -> host

	std::vector<CONTENT_TYPE> result(CONTENT_COUNT);
	std::vector<CONTENT_TYPE> result2(CONTENT_COUNT);
	CUDA_ASSERT(hipMemcpy(result.data(), result_dev, contentTotalBytes, hipMemcpyDeviceToHost));
	CUDA_ASSERT(hipMemcpy(result2.data(), result2_dev, contentTotalBytes, hipMemcpyDeviceToHost));

	puts("Compare results...");
	{
		constexpr uint32_t NUM_SHOW = 8;
		printf("input: [");
		for (size_t i = 0; i < result.size(); ++i) {
			if (i < NUM_SHOW) {
				printf("%d ", input[i]);
			} else if (i == NUM_SHOW) {
				printf("...]\n");
			}
		}
		printf("scan : [");
		for (size_t i = 0; i < result.size(); ++i) {
			CONTENT_TYPE dx = std::abs(answer[i] - result[i]);
			CONTENT_TYPE dx2 = std::abs(answer[i] - result2[i]);
			assert(dx == 0);
			assert(dx2 == 0);
			if (i < NUM_SHOW) {
				printf("%d ", answer[i]);
			} else if (i == NUM_SHOW) {
				printf("...");
			}
		}
		puts("]");
	}

	return 0;
}
