#include "hip/hip_runtime.h"
// Chapter 9. Parallel histogram

#include "tests.cuh"

#include <hip/hip_runtime.h>
#include <>

#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <vector>
#include <array>
#include <algorithm>

// A to Z
#define NUM_CATEGORIES 26

// Naive version that uses atomic operations across all threads.
__global__ void kernel_histogram_naive(
	uint8_t* content, size_t totalLength,
	uint32_t* outHistogram)
{
	int ix = threadIdx.x + (blockDim.x * blockIdx.x);
	if (ix >= totalLength) {
		return;
	}

	int cat = int(content[ix]) - 'a';
	if (0 <= cat && cat < NUM_CATEGORIES) {
		atomicAdd(outHistogram + cat, 1);
	}
}

// Privatization: Contention occurs only between threads in the same block and merging step.
__global__ void kernel_histogram_private(
	uint8_t* content, size_t totalLength,
	uint32_t* outHistogram)
{
	__shared__ uint32_t histo_s[NUM_CATEGORIES];
	if (threadIdx.x < NUM_CATEGORIES) {
		histo_s[threadIdx.x] = 0;
	}
	__syncthreads();

	int ix = threadIdx.x + (blockDim.x * blockIdx.x);
	if (ix >= totalLength) {
		return;
	}

	int cat = int(content[ix]) - 'a';
	if (0 <= cat && cat < NUM_CATEGORIES) {
		atomicAdd(histo_s + cat, 1);
	}
	__syncthreads();

	// Assumes blockDim.x >= NUM_CATEGORIES
	if (threadIdx.x < NUM_CATEGORIES) {
		if (histo_s[threadIdx.x] > 0) {
			atomicAdd(outHistogram + threadIdx.x, histo_s[threadIdx.x]);
		}
	}
}

int runTest_histogram(int argc, char** argv)
{
	// ------------------------------------------
	// Query device properties

	int cudaDeviceId;
	CUDA_ASSERT(hipGetDevice(&cudaDeviceId));

	hipDeviceProp_t deviceProps;
	CUDA_ASSERT(hipGetDeviceProperties(&deviceProps, cudaDeviceId));

	const float KHZ_TO_GHZ = 0.001f * 0.001f;

	puts("CUDA device properties");
	// CUDA gives you all these info!?
	printf("\ttotalConstMem      : %zu bytes\n", deviceProps.totalConstMem);
	printf("\tsharedMemPerBlock  : %zu bytes\n", deviceProps.sharedMemPerBlock);
	printf("\twarpSize           : %d\n", deviceProps.warpSize);
	printf("\tclockRate          : %f GHz\n", KHZ_TO_GHZ * (float)deviceProps.clockRate);
	printf("\tmemoryBusWidth     : %d bits\n", deviceProps.memoryBusWidth);
	printf("\tmemoryClockRate    : %f GHz\n", KHZ_TO_GHZ * deviceProps.memoryClockRate);
	printf("\tmaxThreadsPerBlock : %d\n", deviceProps.maxThreadsPerBlock);

	// ------------------------------------------
	// Read content

	const char* filename = "content/vulkan_handles.hpp";

	printf("Open content file: %s\n", filename);

	FILE* file = fopen(filename, "r");
	if (file == nullptr) {
		printf("Can't find: %s\n", filename);
		puts("Check if current directory is the solution dir");
		return 1;
	}

	fseek(file, 0, SEEK_END);
	size_t fileSize = (size_t)ftell(file);
	fseek(file, 0, SEEK_SET);

	std::vector<uint8_t> content(fileSize, 0);
	fread_s(content.data(), content.size(), sizeof(uint8_t), fileSize, file);

	printf("Content read: %zu bytes\n", fileSize);

	fclose(file);

	// ------------------------------------------
	// Host -> device

	const size_t contentLength = content.size();
	const size_t contentTotalBytes = sizeof(uint8_t) * content.size();
	const size_t categoryTotalBytes = sizeof(uint32_t) * NUM_CATEGORIES;

	uint8_t* content_dev;
	uint32_t* histogram_dev;
	uint32_t* histogram2_dev;
	CUDA_ASSERT(hipMalloc(&content_dev, contentTotalBytes));
	CUDA_ASSERT(hipMalloc(&histogram_dev, categoryTotalBytes));
	CUDA_ASSERT(hipMalloc(&histogram2_dev, categoryTotalBytes));
	CUDA_ASSERT(hipMemcpy(content_dev, content.data(), contentTotalBytes, hipMemcpyHostToDevice));
	CUDA_ASSERT(hipMemset(histogram_dev, 0, categoryTotalBytes));
	CUDA_ASSERT(hipMemset(histogram2_dev, 0, categoryTotalBytes));

	// ------------------------------------------
	// Kernel: histogram

	puts("Run kernel: naive histogram");

	const dim3 blockDim(deviceProps.maxThreadsPerBlock, 1, 1);
	const dim3 numBlocks = calcNumBlocks(dim3(contentLength, 1, 1), blockDim);

	printf("\tblockDim: (%u, %u, %u)\n", blockDim.x, blockDim.y, blockDim.z);
	printf("\tnumBlocks: (%u, %u, %u)\n", numBlocks.x, numBlocks.y, numBlocks.z);

	kernel_histogram_naive<<<numBlocks, blockDim>>>(
		content_dev, contentLength,
		histogram_dev);

	kernel_histogram_private<<<numBlocks, blockDim>>>(
		content_dev, contentLength,
		histogram2_dev);

	// ------------------------------------------
	// Device -> host

	std::array<uint32_t, NUM_CATEGORIES> histogram;
	hipMemcpy(histogram.data(), histogram_dev, categoryTotalBytes, hipMemcpyDeviceToHost);

	std::array<uint32_t, NUM_CATEGORIES> histogram2;
	hipMemcpy(histogram2.data(), histogram2_dev, categoryTotalBytes, hipMemcpyDeviceToHost);

	puts("Compare results...");
	{
		for (size_t i = 0; i < histogram.size(); ++i) {
			assert(histogram[i] == histogram2[i]);
		}
	}

	uint32_t maxCount = 0;
	for (size_t i = 0; i < NUM_CATEGORIES; ++i) {
		printf("(%c, %u) ", (unsigned char)('a' + i), histogram[i]);
		maxCount = std::max(maxCount, histogram[i]);
	}
	puts("");

	for (size_t i = 0; i < NUM_CATEGORIES; ++i) {
		printf("%c", (unsigned char)('a' + i));
	}
	puts("");

	const size_t numRows = 10;
	for (size_t row = 0; row < numRows; ++row) {
		for (size_t i = 0; i < NUM_CATEGORIES; ++i) {
			float ratio = (float)histogram[i] / (float)maxCount;
			if (size_t(ratio * numRows) >= row) {
				printf("*");
			} else {
				printf(" ");
			}
		}
		puts("");
	}

	return 0;
}
